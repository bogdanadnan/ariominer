#include "hip/hip_runtime.h"
#include <hip/driver_types.h>

#include "../../../common/common.h"
#include "../../../app/arguments.h"

#include "../../hasher.h"
#include "../../argon2/argon2.h"

#include "cuda_hasher.h"

#define ITEMS_PER_SEGMENT               32
#define BLOCK_SIZE_UINT4                64
#define BLOCK_SIZE_UINT                256
#define KERNEL_WORKGROUP_SIZE   		32

#define COMPUTE	\
	asm ("{"	\
		".reg .u32 s1, s2, s3, s4;\n\t"	\
		"mul.lo.u32 s3, %0, %2;\n\t"	\
		"mul.hi.u32 s4, %0, %2;\n\t"	\
		"add.cc.u32 s3, s3, s3;\n\t"	\
		"addc.u32 s4, s4, s4;\n\t"	\
		"add.cc.u32 s1, %0, %2;\n\t"	\
		"addc.u32 s2, %1, %3;\n\t"	\
		"add.cc.u32 %0, s1, s3;\n\t"	\
		"addc.u32 %1, s2, s4;\n\t"	\
		"xor.b32 s1, %0, %6;\n\t"	\
		"xor.b32 %6, %1, %7;\n\t"	\
		"mov.b32 %7, s1;\n\t"	\
		"mul.lo.u32 s3, %4, %6;\n\t"	\
		"mul.hi.u32 s4, %4, %6;\n\t"	\
		"add.cc.u32 s3, s3, s3;\n\t"	\
		"addc.u32 s4, s4, s4;\n\t"	\
		"add.cc.u32 s1, %4, %6;\n\t"	\
		"addc.u32 s2, %5, %7;\n\t"	\
		"add.cc.u32 %4, s1, s3;\n\t"	\
		"addc.u32 %5, s2, s4;\n\t"	\
		"xor.b32 s3, %2, %4;\n\t"	\
		"xor.b32 s4, %3, %5;\n\t"	\
		"shf.r.wrap.b32 %3, s4, s3, 24;\n\t"	\
		"shf.r.wrap.b32 %2, s3, s4, 24;\n\t"	\
		"mul.lo.u32 s3, %0, %2;\n\t"	\
		"mul.hi.u32 s4, %0, %2;\n\t"	\
		"add.cc.u32 s3, s3, s3;\n\t"	\
		"addc.u32 s4, s4, s4;\n\t"	\
		"add.cc.u32 s1, %0, %2;\n\t"	\
		"addc.u32 s2, %1, %3;\n\t"	\
		"add.cc.u32 %0, s1, s3;\n\t"	\
		"addc.u32 %1, s2, s4;\n\t"	\
		"xor.b32 s3, %0, %6;\n\t"	\
		"xor.b32 s4, %1, %7;\n\t"	\
		"shf.r.wrap.b32 %7, s4, s3, 16;\n\t"	\
		"shf.r.wrap.b32 %6, s3, s4, 16;\n\t"	\
		"mul.lo.u32 s3, %4, %6;\n\t"	\
		"mul.hi.u32 s4, %4, %6;\n\t"	\
		"add.cc.u32 s3, s3, s3;\n\t"	\
		"addc.u32 s4, s4, s4;\n\t"	\
		"add.cc.u32 s1, %4, %6;\n\t"	\
		"addc.u32 s2, %5, %7;\n\t"	\
		"add.cc.u32 %4, s1, s3;\n\t"	\
		"addc.u32 %5, s2, s4;\n\t"	\
		"xor.b32 s3, %2, %4;\n\t"	\
		"xor.b32 s4, %3, %5;\n\t"	\
		"shf.r.wrap.b32 %3, s3, s4, 31;\n\t"	\
		"shf.r.wrap.b32 %2, s4, s3, 31;\n\t"	\
	"}" : "+r"(tmp_a.x), "+r"(tmp_a.y), "+r"(tmp_a.z), "+r"(tmp_a.w), "+r"(tmp_b.x), "+r"(tmp_b.y), "+r"(tmp_b.z), "+r"(tmp_b.w));

#define G1(data)           \
{                           \
	COMPUTE \
	tmp_a.z = __shfl_sync(0xffffffff, tmp_a.z, i_shfl1_1); \
	tmp_a.w = __shfl_sync(0xffffffff, tmp_a.w, i_shfl1_1); \
	tmp_b.x = __shfl_sync(0xffffffff, tmp_b.x, i_shfl1_2); \
	tmp_b.y = __shfl_sync(0xffffffff, tmp_b.y, i_shfl1_2); \
	tmp_b.z = __shfl_sync(0xffffffff, tmp_b.z, i_shfl1_3); \
	tmp_b.w = __shfl_sync(0xffffffff, tmp_b.w, i_shfl1_3); \
}

#define G2(data)           \
{ \
	COMPUTE \
    data[i2_0_0] = tmp_a.x; \
    data[i2_0_1] = tmp_a.y; \
    data[i2_1_0] = tmp_a.z; \
    data[i2_1_1] = tmp_a.w; \
    data[i2_2_0] = tmp_b.x; \
    data[i2_2_1] = tmp_b.y; \
    data[i2_3_0] = tmp_b.z; \
    data[i2_3_1] = tmp_b.w; \
    __syncwarp(); \
}

#define G3(data)           \
{                           \
    tmp_a.x = data[i3_0_0]; \
    tmp_a.y = data[i3_0_1]; \
    tmp_a.z = data[i3_1_0]; \
    tmp_a.w = data[i3_1_1]; \
    tmp_b.x = data[i3_2_0]; \
    tmp_b.y = data[i3_2_1]; \
    tmp_b.z = data[i3_3_0]; \
    tmp_b.w = data[i3_3_1]; \
	COMPUTE \
	tmp_a.z = __shfl_sync(0xffffffff, tmp_a.z, i_shfl2_1); \
	tmp_a.w = __shfl_sync(0xffffffff, tmp_a.w, i_shfl2_1); \
	tmp_b.x = __shfl_sync(0xffffffff, tmp_b.x, i_shfl2_2); \
	tmp_b.y = __shfl_sync(0xffffffff, tmp_b.y, i_shfl2_2); \
	tmp_b.z = __shfl_sync(0xffffffff, tmp_b.z, i_shfl2_3); \
	tmp_b.w = __shfl_sync(0xffffffff, tmp_b.w, i_shfl2_3); \
}

#define G4(data)           \
{                           \
	COMPUTE \
    data[i4_0_0] = tmp_a.x; \
    data[i4_0_1] = tmp_a.y; \
    data[i4_1_0] = tmp_a.z; \
    data[i4_1_1] = tmp_a.w; \
    data[i4_2_0] = tmp_b.x; \
    data[i4_2_1] = tmp_b.y; \
    data[i4_3_0] = tmp_b.z; \
    data[i4_3_1] = tmp_b.w; \
    __syncwarp(); \
    tmp_a.x = data[i1_0_0]; \
    tmp_a.y = data[i1_0_1]; \
    tmp_a.z = data[i1_1_0]; \
    tmp_a.w = data[i1_1_1]; \
    tmp_b.x = data[i1_2_0]; \
    tmp_b.y = data[i1_2_1]; \
    tmp_b.z = data[i1_3_0]; \
    tmp_b.w = data[i1_3_1]; \
}

__constant__ int offsets[768] = {
		0, 4, 8, 12,
		1, 5, 9, 13,
		2, 6, 10, 14,
		3, 7, 11, 15,
		16, 20, 24, 28,
		17, 21, 25, 29,
		18, 22, 26, 30,
		19, 23, 27, 31,
		32, 36, 40, 44,
		33, 37, 41, 45,
		34, 38, 42, 46,
		35, 39, 43, 47,
		48, 52, 56, 60,
		49, 53, 57, 61,
		50, 54, 58, 62,
		51, 55, 59, 63,
		64, 68, 72, 76,
		65, 69, 73, 77,
		66, 70, 74, 78,
		67, 71, 75, 79,
		80, 84, 88, 92,
		81, 85, 89, 93,
		82, 86, 90, 94,
		83, 87, 91, 95,
		96, 100, 104, 108,
		97, 101, 105, 109,
		98, 102, 106, 110,
		99, 103, 107, 111,
		112, 116, 120, 124,
		113, 117, 121, 125,
		114, 118, 122, 126,
		115, 119, 123, 127,
		0, 5, 10, 15,
		1, 6, 11, 12,
		2, 7, 8, 13,
		3, 4, 9, 14,
		16, 21, 26, 31,
		17, 22, 27, 28,
		18, 23, 24, 29,
		19, 20, 25, 30,
		32, 37, 42, 47,
		33, 38, 43, 44,
		34, 39, 40, 45,
		35, 36, 41, 46,
		48, 53, 58, 63,
		49, 54, 59, 60,
		50, 55, 56, 61,
		51, 52, 57, 62,
		64, 69, 74, 79,
		65, 70, 75, 76,
		66, 71, 72, 77,
		67, 68, 73, 78,
		80, 85, 90, 95,
		81, 86, 91, 92,
		82, 87, 88, 93,
		83, 84, 89, 94,
		96, 101, 106, 111,
		97, 102, 107, 108,
		98, 103, 104, 109,
		99, 100, 105, 110,
		112, 117, 122, 127,
		113, 118, 123, 124,
		114, 119, 120, 125,
		115, 116, 121, 126,
		0, 32, 64, 96,
		1, 33, 65, 97,
		2, 34, 66, 98,
		3, 35, 67, 99,
		4, 36, 68, 100,
		5, 37, 69, 101,
		6, 38, 70, 102,
		7, 39, 71, 103,
		8, 40, 72, 104,
		9, 41, 73, 105,
		10, 42, 74, 106,
		11, 43, 75, 107,
		12, 44, 76, 108,
		13, 45, 77, 109,
		14, 46, 78, 110,
		15, 47, 79, 111,
		16, 48, 80, 112,
		17, 49, 81, 113,
		18, 50, 82, 114,
		19, 51, 83, 115,
		20, 52, 84, 116,
		21, 53, 85, 117,
		22, 54, 86, 118,
		23, 55, 87, 119,
		24, 56, 88, 120,
		25, 57, 89, 121,
		26, 58, 90, 122,
		27, 59, 91, 123,
		28, 60, 92, 124,
		29, 61, 93, 125,
		30, 62, 94, 126,
		31, 63, 95, 127,
		0, 33, 80, 113,
		1, 48, 81, 96,
		2, 35, 82, 115,
		3, 50, 83, 98,
		4, 37, 84, 117,
		5, 52, 85, 100,
		6, 39, 86, 119,
		7, 54, 87, 102,
		8, 41, 88, 121,
		9, 56, 89, 104,
		10, 43, 90, 123,
		11, 58, 91, 106,
		12, 45, 92, 125,
		13, 60, 93, 108,
		14, 47, 94, 127,
		15, 62, 95, 110,
		16, 49, 64, 97,
		17, 32, 65, 112,
		18, 51, 66, 99,
		19, 34, 67, 114,
		20, 53, 68, 101,
		21, 36, 69, 116,
		22, 55, 70, 103,
		23, 38, 71, 118,
		24, 57, 72, 105,
		25, 40, 73, 120,
		26, 59, 74, 107,
		27, 42, 75, 122,
		28, 61, 76, 109,
		29, 44, 77, 124,
		30, 63, 78, 111,
		31, 46, 79, 126,
        0, 1, 2, 3,
        1, 2, 3, 0,
        2, 3, 0, 1,
        3, 0, 1, 2,
        4, 5, 6, 7,
        5, 6, 7, 4,
        6, 7, 4, 5,
        7, 4, 5, 6,
        8, 9, 10, 11,
        9, 10, 11, 8,
        10, 11, 8, 9,
        11, 8, 9, 10,
        12, 13, 14, 15,
        13, 14, 15, 12,
        14, 15, 12, 13,
        15, 12, 13, 14,
        16, 17, 18, 19,
        17, 18, 19, 16,
        18, 19, 16, 17,
        19, 16, 17, 18,
        20, 21, 22, 23,
        21, 22, 23, 20,
        22, 23, 20, 21,
        23, 20, 21, 22,
        24, 25, 26, 27,
        25, 26, 27, 24,
        26, 27, 24, 25,
        27, 24, 25, 26,
        28, 29, 30, 31,
        29, 30, 31, 28,
        30, 31, 28, 29,
        31, 28, 29, 30,
        0, 1, 16, 17,
        1, 16, 17, 0,
        2, 3, 18, 19,
        3, 18, 19, 2,
        4, 5, 20, 21,
        5, 20, 21, 4,
        6, 7, 22, 23,
        7, 22, 23, 6,
        8, 9, 24, 25,
        9, 24, 25, 8,
        10, 11, 26, 27,
        11, 26, 27, 10,
        12, 13, 28, 29,
        13, 28, 29, 12,
        14, 15, 30, 31,
        15, 30, 31, 14,
        16, 17, 0, 1,
        17, 0, 1, 16,
        18, 19, 2, 3,
        19, 2, 3, 18,
        20, 21, 4, 5,
        21, 4, 5, 20,
        22, 23, 6, 7,
        23, 6, 7, 22,
        24, 25, 8, 9,
        25, 8, 9, 24,
        26, 27, 10, 11,
        27, 10, 11, 26,
        28, 29, 12, 13,
        29, 12, 13, 28,
        30, 31, 14, 15,
        31, 14, 15, 30
};

inline __host__ __device__ void operator^=( uint4& a, uint4 s) {
   a.x ^= s.x; a.y ^= s.y; a.z ^= s.z; a.w ^= s.w;
}

__global__ void fill_blocks_cpu(uint32_t *scratchpad0,
                                uint32_t *scratchpad1,
                                uint32_t *scratchpad2,
                                uint32_t *scratchpad3,
                                uint32_t *scratchpad4,
                                uint32_t *scratchpad5,
                                uint32_t *seed,
                                uint32_t *out,
                                int32_t *addresses,
                                int memsize,
                                int threads_per_chunk,
								int thread_idx) {
	__shared__ uint32_t state[BLOCK_SIZE_UINT];
	__shared__ int32_t addr[64];

	uint4 tmp_a, tmp_b, tmp_c, tmp_d, tmp_p, tmp_q;

	int hash = blockIdx.x;
    int mem_hash = hash + thread_idx;
	int id = threadIdx.x;

	int offset = id << 2;

	int i1_0_0 = 2 * offsets[offset];
	int i1_0_1 = i1_0_0 + 1;
	int i1_1_0 = 2 * offsets[offset + 1];
    int i1_1_1 = i1_1_0 + 1;
	int i1_2_0 = 2 * offsets[offset + 2];
    int i1_2_1 = i1_2_0 + 1;
	int i1_3_0 = 2 * offsets[offset + 3];
    int i1_3_1 = i1_3_0 + 1;

	int i2_0_0 = 2 * offsets[offset + 128];
    int i2_0_1 = i2_0_0 + 1;
	int i2_1_0 = 2 * offsets[offset + 129];
    int i2_1_1 = i2_1_0 + 1;
	int i2_2_0 = 2 * offsets[offset + 130];
    int i2_2_1 = i2_2_0 + 1;
	int i2_3_0 = 2 * offsets[offset + 131];
    int i2_3_1 = i2_3_0 + 1;

	int i3_0_0 = 2 * offsets[offset + 256];
    int i3_0_1 = i3_0_0 + 1;
	int i3_1_0 = 2 * offsets[offset + 257];
    int i3_1_1 = i3_1_0 + 1;
	int i3_2_0 = 2 * offsets[offset + 258];
    int i3_2_1 = i3_2_0 + 1;
	int i3_3_0 = 2 * offsets[offset + 259];
    int i3_3_1 = i3_3_0 + 1;

	int i4_0_0 = 2 * offsets[offset + 384];
    int i4_0_1 = i4_0_0 + 1;
	int i4_1_0 = 2 * offsets[offset + 385];
    int i4_1_1 = i4_1_0 + 1;
	int i4_2_0 = 2 * offsets[offset + 386];
    int i4_2_1 = i4_2_0 + 1;
	int i4_3_0 = 2 * offsets[offset + 387];
    int i4_3_1 = i4_3_0 + 1;

	int i_shfl1_1 = offsets[offset + 513];
	int i_shfl1_2 = offsets[offset + 514];
	int i_shfl1_3 = offsets[offset + 515];
	int i_shfl2_1 = offsets[offset + 641];
	int i_shfl2_2 = offsets[offset + 642];
	int i_shfl2_3 = offsets[offset + 643];

    int scratchpad_location = mem_hash / threads_per_chunk;
    uint4 *memory = reinterpret_cast<uint4*>(scratchpad0);
    if(scratchpad_location == 1) memory = reinterpret_cast<uint4*>(scratchpad1);
    if(scratchpad_location == 2) memory = reinterpret_cast<uint4*>(scratchpad2);
    if(scratchpad_location == 3) memory = reinterpret_cast<uint4*>(scratchpad3);
    if(scratchpad_location == 4) memory = reinterpret_cast<uint4*>(scratchpad4);
    if(scratchpad_location == 5) memory = reinterpret_cast<uint4*>(scratchpad5);
    int hash_offset = mem_hash - scratchpad_location * threads_per_chunk;
    memory = memory + hash_offset * (memsize >> 4);

	uint32_t *seed_src = seed + hash * 2 * BLOCK_SIZE_UINT;

	uint4 *seed_dst = memory;
	seed_dst[id] = make_uint4(seed_src[i1_0_0], seed_src[i1_0_1], seed_src[i1_1_0], seed_src[i1_1_1]);
	seed_dst[id + 32] = make_uint4(seed_src[i1_2_0], seed_src[i1_2_1], seed_src[i1_3_0], seed_src[i1_3_1]);
	seed_src += BLOCK_SIZE_UINT;
	seed_dst += BLOCK_SIZE_UINT4;
    tmp_a = make_uint4(seed_src[i1_0_0], seed_src[i1_0_1], seed_src[i1_1_0], seed_src[i1_1_1]);
    tmp_b = make_uint4(seed_src[i1_2_0], seed_src[i1_2_1], seed_src[i1_3_0], seed_src[i1_3_1]);
	seed_dst[id] = tmp_a;
	seed_dst[id + 32] = tmp_b;

	uint4 *next_block;
	uint4 *ref_block;

	int32_t *stop_addr = addresses + 524286 * 2;

	for(; addresses < stop_addr; addresses += 64) {
		addr[id] = addresses[id];
		addr[id + 32] = addresses[id + 32];

		uint64_t i_limit = (stop_addr - addresses) >> 1;
		if(i_limit > 32) i_limit = 32;

		int32_t addr_n = 0;
		int32_t addr0 = addr[0];
		int32_t addr1 = addr[32];
		ref_block = memory + addr1 * BLOCK_SIZE_UINT4;
        tmp_p = ref_block[id];
        tmp_q = ref_block[id + 32];

		for(int i=0;i<i_limit;i++) {
			addr_n = addr0;
			tmp_a ^= tmp_p; tmp_b ^= tmp_q;

			if (i < i_limit - 1) {
				addr0 = addr[i + 1];
				addr1 = addr[i + 33];

				ref_block = memory + addr1 * BLOCK_SIZE_UINT4;
                tmp_p = ref_block[id];
                tmp_q = ref_block[id + 32];
			}

			tmp_c = tmp_a; tmp_d = tmp_b;

			G1(state);
			G2(state);
			G3(state);
			G4(state);

			tmp_a ^= tmp_c; tmp_b ^= tmp_d;

			if (addr_n != -1) {
				next_block = memory + addr_n * BLOCK_SIZE_UINT4;
                next_block[id] = tmp_a;
                next_block[id + 32] = tmp_b;
			}
		}
	}

	uint32_t *out_mem = out + hash * 2 * BLOCK_SIZE_UINT;
    out_mem[i1_0_0] = tmp_a.x;
    out_mem[i1_0_1] = tmp_a.y;
    out_mem[i1_1_0] = tmp_a.z;
    out_mem[i1_1_1] = tmp_a.w;
    out_mem[i1_2_0] = tmp_b.x;
    out_mem[i1_2_1] = tmp_b.y;
    out_mem[i1_3_0] = tmp_b.z;
    out_mem[i1_3_1] = tmp_b.w;
};

__global__ void fill_blocks_gpu(uint32_t *scratchpad0,
                                uint32_t *scratchpad1,
                                uint32_t *scratchpad2,
                                uint32_t *scratchpad3,
                                uint32_t *scratchpad4,
                                uint32_t *scratchpad5,
                                uint32_t *seed,
                                uint32_t *out,
                                uint32_t *addresses,
                                uint32_t *segments,
                                int memsize,
                                int threads_per_chunk,
								int thread_idx) {
	__shared__ uint32_t state[4 * BLOCK_SIZE_UINT];
	__shared__ uint32_t addr[4 * 32];

	uint4 tmp_a, tmp_b, tmp_c, tmp_d, tmp_p, tmp_q, tmp_l, tmp_m;

	int hash = blockIdx.x;
	int mem_hash = hash + thread_idx;
	int local_id = threadIdx.x;

	int id = local_id % ITEMS_PER_SEGMENT;
	int segment = local_id / ITEMS_PER_SEGMENT;

	int offset = id << 2;

	int i1_0_0 = 2 * offsets[offset];
	int i1_0_1 = i1_0_0 + 1;
	int i1_1_0 = 2 * offsets[offset + 1];
	int i1_1_1 = i1_1_0 + 1;
	int i1_2_0 = 2 * offsets[offset + 2];
	int i1_2_1 = i1_2_0 + 1;
	int i1_3_0 = 2 * offsets[offset + 3];
	int i1_3_1 = i1_3_0 + 1;

	int i2_0_0 = 2 * offsets[offset + 128];
	int i2_0_1 = i2_0_0 + 1;
	int i2_1_0 = 2 * offsets[offset + 129];
	int i2_1_1 = i2_1_0 + 1;
	int i2_2_0 = 2 * offsets[offset + 130];
	int i2_2_1 = i2_2_0 + 1;
	int i2_3_0 = 2 * offsets[offset + 131];
	int i2_3_1 = i2_3_0 + 1;

	int i3_0_0 = 2 * offsets[offset + 256];
	int i3_0_1 = i3_0_0 + 1;
	int i3_1_0 = 2 * offsets[offset + 257];
	int i3_1_1 = i3_1_0 + 1;
	int i3_2_0 = 2 * offsets[offset + 258];
	int i3_2_1 = i3_2_0 + 1;
	int i3_3_0 = 2 * offsets[offset + 259];
	int i3_3_1 = i3_3_0 + 1;

	int i4_0_0 = 2 * offsets[offset + 384];
	int i4_0_1 = i4_0_0 + 1;
	int i4_1_0 = 2 * offsets[offset + 385];
	int i4_1_1 = i4_1_0 + 1;
	int i4_2_0 = 2 * offsets[offset + 386];
	int i4_2_1 = i4_2_0 + 1;
	int i4_3_0 = 2 * offsets[offset + 387];
	int i4_3_1 = i4_3_0 + 1;

	int i_shfl1_1 = offsets[offset + 513];
	int i_shfl1_2 = offsets[offset + 514];
	int i_shfl1_3 = offsets[offset + 515];
	int i_shfl2_1 = offsets[offset + 641];
	int i_shfl2_2 = offsets[offset + 642];
	int i_shfl2_3 = offsets[offset + 643];

    int scratchpad_location = mem_hash / threads_per_chunk;
    uint4 *memory = reinterpret_cast<uint4*>(scratchpad0);
    if(scratchpad_location == 1) memory = reinterpret_cast<uint4*>(scratchpad1);
    if(scratchpad_location == 2) memory = reinterpret_cast<uint4*>(scratchpad2);
    if(scratchpad_location == 3) memory = reinterpret_cast<uint4*>(scratchpad3);
    if(scratchpad_location == 4) memory = reinterpret_cast<uint4*>(scratchpad4);
    if(scratchpad_location == 5) memory = reinterpret_cast<uint4*>(scratchpad5);
    int hash_offset = mem_hash - scratchpad_location * threads_per_chunk;
    memory = memory + hash_offset * (memsize >> 4);

	uint32_t *mem_seed = seed + hash * 8 * BLOCK_SIZE_UINT;

	uint32_t *seed_src = mem_seed + segment * 2 * BLOCK_SIZE_UINT;
	uint4 *seed_dst = memory + segment * 4096 * BLOCK_SIZE_UINT4;

	seed_dst[id] = make_uint4(seed_src[i1_0_0], seed_src[i1_0_1], seed_src[i1_1_0], seed_src[i1_1_1]);
	seed_dst[id + 32] = make_uint4(seed_src[i1_2_0], seed_src[i1_2_1], seed_src[i1_3_0], seed_src[i1_3_1]);
	seed_src += BLOCK_SIZE_UINT;
	seed_dst += BLOCK_SIZE_UINT4;
	seed_dst[id] = make_uint4(seed_src[i1_0_0], seed_src[i1_0_1], seed_src[i1_1_0], seed_src[i1_1_1]);
	seed_dst[id + 32] = make_uint4(seed_src[i1_2_0], seed_src[i1_2_1], seed_src[i1_3_0], seed_src[i1_3_1]);

	uint4 *next_block;
	uint4 *prev_block;
	uint4 *ref_block;

	uint32_t *local_state = state + segment * BLOCK_SIZE_UINT;
	uint32_t *local_addr = addr + segment * 32;

	segments += segment;
	uint16_t addr_start_idx = 0;
	uint16_t prev_blk_idx;
	int inc = 1022;

	//without xor
	for(int s=0; s<4; s++) {
		uint32_t curr_seg = segments[s * 4];

		asm("mov.b32 {%0, %1}, %2;"
		: "=h"(addr_start_idx), "=h"(prev_blk_idx) : "r"(curr_seg));

		uint32_t *addr = addresses + addr_start_idx;
		uint32_t *stop_addr = addresses + addr_start_idx + inc;
		inc = 1024;

		prev_block = memory + prev_blk_idx * BLOCK_SIZE_UINT4;
		__syncthreads();

		tmp_a = prev_block[id];
		tmp_b = prev_block[id + 32];

		for(; addr < stop_addr; addr += 32) {
			local_addr[id] = addr[id];

			uint64_t i_limit = stop_addr - addr;
			if(i_limit > 32) i_limit = 32;

			int16_t addr0, addr1;
			asm("{mov.b32 {%0, %1}, %2;}": "=h"(addr0), "=h"(addr1) : "r"(local_addr[0]));
			ref_block = memory + addr1 * BLOCK_SIZE_UINT4;
			tmp_p = ref_block[id];
			tmp_q = ref_block[id + 32];

			for(int i=0;i<i_limit;i++) {
				next_block = memory + addr0 * BLOCK_SIZE_UINT4;

                tmp_a ^= tmp_p; tmp_b ^= tmp_q;

				if (i < (i_limit - 1)) {
					asm("{mov.b32 {%0, %1}, %2;}": "=h"(addr0), "=h"(addr1) : "r"(local_addr[i + 1]));
					ref_block = memory + addr1 * BLOCK_SIZE_UINT4;
                    tmp_p = ref_block[id];
                    tmp_q = ref_block[id + 32];
				}

				tmp_c = tmp_a; tmp_d = tmp_b;

				G1(local_state);
				G2(local_state);
				G3(local_state);
				G4(local_state);

				tmp_a ^= tmp_c; tmp_b ^= tmp_d;

				next_block[id] = tmp_a;
				next_block[id + 32] = tmp_b;
			}
		}
	}

	// with xor
	for(int s=4; s<16; s++) {
		uint32_t curr_seg = segments[s * 4];

		asm("mov.b32 {%0, %1}, %2;"
		: "=h"(addr_start_idx), "=h"(prev_blk_idx) : "r"(curr_seg));

		uint32_t *addr = addresses + addr_start_idx;
		uint32_t *stop_addr = addresses + addr_start_idx + 1024;

		prev_block = memory + prev_blk_idx * BLOCK_SIZE_UINT4;
		__syncthreads();

		tmp_a = prev_block[id];
		tmp_b = prev_block[id + 32];

		for(; addr < stop_addr; addr += 32) {
			local_addr[id] = addr[id];

			uint16_t addr0, addr1, dont_store;
			asm("{mov.b32 {%0, %1}, %2;}": "=h"(addr0), "=h"(addr1) : "r"(local_addr[0]));
			ref_block = memory + addr1 * BLOCK_SIZE_UINT4;

            tmp_p = ref_block[id];
			tmp_q = ref_block[id + 32];

			for (int i = 0; i < 32; i++) {
				if(s >= 12) {
					dont_store = addr0 >> 15;
					addr0 = addr0 & 32767;
				}
				else {
					dont_store = 0;
				}
				next_block = memory + addr0 * BLOCK_SIZE_UINT4;

				tmp_l = next_block[id];
				tmp_m = next_block[id + 32];

                tmp_a ^= tmp_p; tmp_b ^= tmp_q;

				if (i < 31) {
					asm("{mov.b32 {%0, %1}, %2;}": "=h"(addr0), "=h"(addr1) : "r"(local_addr[i + 1]));
					ref_block = memory + addr1 * BLOCK_SIZE_UINT4;
                    tmp_p = ref_block[id];
                    tmp_q = ref_block[id + 32];
				}

				tmp_c = tmp_a; tmp_d = tmp_b;

				G1(local_state);
				G2(local_state);
				G3(local_state);
				G4(local_state);

				tmp_c ^= tmp_l; tmp_d ^= tmp_m;
				tmp_a ^= tmp_c; tmp_b ^= tmp_d;

                if(!dont_store) {
                    next_block[id] = tmp_a;
                    next_block[id + 32] = tmp_b;
				}
			}
		}
	}

	__syncthreads();

	int dst_addr = 65528;

	offset = ((segment / 2) << 7) + (id << 2) + ((segment % 2) << 1);
	uint32_t *block = reinterpret_cast<uint32_t*>(memory) + ((int16_t*)(&addresses[dst_addr]))[0] * BLOCK_SIZE_UINT;

    uint32_t data0 = block[offset];
    uint32_t data1 = block[offset + 1];

	for(;dst_addr < 65531; ++dst_addr) {
		block = reinterpret_cast<uint32_t*>(memory) + ((int16_t*)(&addresses[dst_addr]))[1] * BLOCK_SIZE_UINT;
        data0 ^= block[offset];
        data1 ^= block[offset + 1];
	}

	int idx0 = i1_0_0; int idx1 = i1_0_1;
	if (segment == 1) { idx0 = i1_1_0; idx1 = i1_1_1; }
	if (segment == 2) { idx0 = i1_2_0; idx1 = i1_2_1; }
	if (segment == 3) { idx0 = i1_3_0; idx1 = i1_3_1; }

	uint32_t *out_mem = out + hash * 8 * BLOCK_SIZE_UINT;
    out_mem[idx0] = data0;
    out_mem[idx1] = data1;
};

void cuda_allocate(cuda_device_info *device, double chunks, size_t chunk_size) {
	device->error = hipSetDevice(device->cuda_index);
	if(device->error != hipSuccess) {
		device->error_message = "Error setting current device for memory allocation.";
		return;
	}

	size_t allocated_mem_for_current_chunk = 0;

	if (chunks > 0) {
		allocated_mem_for_current_chunk = chunks > 1 ? chunk_size : (size_t)ceil(chunk_size * chunks);
		chunks -= 1;
	}
	else {
		allocated_mem_for_current_chunk = 1;
	}
	device->error = hipMalloc(&device->arguments.memory_chunk_0, allocated_mem_for_current_chunk);
	if (device->error != hipSuccess) {
		device->error_message = "Error allocating memory.";
		return;
	}
	if (chunks > 0) {
		allocated_mem_for_current_chunk = chunks > 1 ? chunk_size : (size_t)ceil(chunk_size * chunks);
		chunks -= 1;
	}
	else {
		allocated_mem_for_current_chunk = 1;
	}
	device->error = hipMalloc(&device->arguments.memory_chunk_1, allocated_mem_for_current_chunk);
	if (device->error != hipSuccess) {
		device->error_message = "Error allocating memory.";
		return;
	}
	if (chunks > 0) {
		allocated_mem_for_current_chunk = chunks > 1 ? chunk_size : (size_t)ceil(chunk_size * chunks);
		chunks -= 1;
	}
	else {
		allocated_mem_for_current_chunk = 1;
	}
	device->error = hipMalloc(&device->arguments.memory_chunk_2, allocated_mem_for_current_chunk);
	if (device->error != hipSuccess) {
		device->error_message = "Error allocating memory.";
		return;
	}
	if (chunks > 0) {
		allocated_mem_for_current_chunk = chunks > 1 ? chunk_size : (size_t)ceil(chunk_size * chunks);
		chunks -= 1;
	}
	else {
		allocated_mem_for_current_chunk = 1;
	}
	device->error = hipMalloc(&device->arguments.memory_chunk_3, allocated_mem_for_current_chunk);
	if (device->error != hipSuccess) {
		device->error_message = "Error allocating memory.";
		return;
	}
	if (chunks > 0) {
		allocated_mem_for_current_chunk = chunks > 1 ? chunk_size : (size_t)ceil(chunk_size * chunks);
		chunks -= 1;
	}
	else {
		allocated_mem_for_current_chunk = 1;
	}
	device->error = hipMalloc(&device->arguments.memory_chunk_4, allocated_mem_for_current_chunk);
	if (device->error != hipSuccess) {
		device->error_message = "Error allocating memory.";
		return;
	}
	if (chunks > 0) {
		allocated_mem_for_current_chunk = chunks > 1 ? chunk_size : (size_t)ceil(chunk_size * chunks);
		chunks -= 1;
	}
	else {
		allocated_mem_for_current_chunk = 1;
	}
	device->error = hipMalloc(&device->arguments.memory_chunk_5, allocated_mem_for_current_chunk);
	if (device->error != hipSuccess) {
		device->error_message = "Error allocating memory.";
		return;
	}

	//optimise address sizes
	int32_t *addresses_1_1_524288 = (int32_t *)malloc((argon2profile_1_1_524288.block_refs_size + 2) * 2 * sizeof(int32_t)); //add 2 to ref_size to be exact multiple of 32

	for(int i=0;i<argon2profile_1_1_524288.block_refs_size;i++) {
		int ref_chunk_idx = (i / 32) * 64;
		int ref_idx = i % 32;

		addresses_1_1_524288[ref_chunk_idx + ref_idx] = argon2profile_1_1_524288.block_refs[i*4];
		addresses_1_1_524288[ref_chunk_idx + ref_idx + 32] = argon2profile_1_1_524288.block_refs[i*4 + 2];
	}
	device->error = hipMalloc(&device->arguments.address_profile_1_1_524288, (argon2profile_1_1_524288.block_refs_size + 2) * 2 * sizeof(int32_t));
	if(device->error != hipSuccess) {
		device->error_message = "Error allocating memory.";
		return;
	}
	device->error = hipMemcpy(device->arguments.address_profile_1_1_524288, addresses_1_1_524288, (argon2profile_1_1_524288.block_refs_size + 2) * 2 * sizeof(int32_t), hipMemcpyHostToDevice);
	if(device->error != hipSuccess) {
		device->error_message = "Error copying memory.";
		return;
	}
	free(addresses_1_1_524288);

	//optimise address sizes
	uint16_t *addresses_4_4_16384 = (uint16_t *)malloc(argon2profile_4_4_16384.block_refs_size * 2 * sizeof(uint16_t));
	for(int i=0;i<argon2profile_4_4_16384.block_refs_size;i++) {
		addresses_4_4_16384[i*2] = argon2profile_4_4_16384.block_refs[i*4 + (i >= 65528 ? 1 : 0)];
		addresses_4_4_16384[i*2 + 1] = argon2profile_4_4_16384.block_refs[i*4 + 2];
		if(argon2profile_4_4_16384.block_refs[i*4 + 3] == 0) {
            addresses_4_4_16384[i*2] |= 32768;
		}
	}
	device->error = hipMalloc(&device->arguments.address_profile_4_4_16384, argon2profile_4_4_16384.block_refs_size * 2 * sizeof(uint16_t));
	if(device->error != hipSuccess) {
		device->error_message = "Error allocating memory.";
		return;
	}
	device->error = hipMemcpy(device->arguments.address_profile_4_4_16384, addresses_4_4_16384, argon2profile_4_4_16384.block_refs_size * 2 * sizeof(uint16_t), hipMemcpyHostToDevice);
	if(device->error != hipSuccess) {
		device->error_message = "Error copying memory.";
		return;
	}
	free(addresses_4_4_16384);

	//reorganize segments data
	uint16_t *segments_4_4_16384 = (uint16_t *)malloc(64 * 2 * sizeof(uint16_t));
	for(int i=0;i<64;i++) {
		int seg_start = argon2profile_4_4_16384.segments[i*3];
		segments_4_4_16384[i*2] = seg_start;
		segments_4_4_16384[i*2 + 1] = argon2profile_4_4_16384.block_refs[seg_start*4 + 1];
	}
	device->error = hipMalloc(&device->arguments.segments_profile_4_4_16384, 64 * 2 * sizeof(uint16_t));
	if(device->error != hipSuccess) {
		device->error_message = "Error allocating memory.";
		return;
	}
	device->error = hipMemcpy(device->arguments.segments_profile_4_4_16384, segments_4_4_16384, 64 * 2 * sizeof(uint16_t), hipMemcpyHostToDevice);
	if(device->error != hipSuccess) {
		device->error_message = "Error copying memory.";
		return;
	}
	free(segments_4_4_16384);

    size_t max_threads = max(device->profile_info.threads_profile_4_4_16384, device->profile_info.threads_profile_1_1_524288);
    size_t accessory_memory_size = max_threads * 8 * ARGON2_BLOCK_SIZE;
    device->error = hipMalloc(&device->arguments.seed_memory[0], accessory_memory_size);
    if (device->error != hipSuccess) {
        device->error_message = "Error allocating memory.";
        return;
    }
    device->error = hipMalloc(&device->arguments.out_memory[0], accessory_memory_size);
    if (device->error != hipSuccess) {
        device->error_message = "Error allocating memory.";
        return;
    }
    device->error = hipHostMalloc(&device->arguments.host_seed_memory[0], accessory_memory_size);
    if (device->error != hipSuccess) {
        device->error_message = "Error allocating pinned memory.";
        return;
    }
    device->error = hipMalloc(&device->arguments.seed_memory[1], accessory_memory_size);
    if (device->error != hipSuccess) {
        device->error_message = "Error allocating memory.";
        return;
    }
    device->error = hipMalloc(&device->arguments.out_memory[1], accessory_memory_size);
    if (device->error != hipSuccess) {
        device->error_message = "Error allocating memory.";
        return;
    }
    device->error = hipHostMalloc(&device->arguments.host_seed_memory[1], accessory_memory_size);
    if (device->error != hipSuccess) {
        device->error_message = "Error allocating pinned memory.";
        return;
    }
}

void cuda_free(cuda_device_info *device) {
	hipSetDevice(device->cuda_index);

	if(device->arguments.address_profile_1_1_524288 != NULL) {
		hipFree(device->arguments.address_profile_1_1_524288);
		device->arguments.address_profile_1_1_524288 = NULL;
	}

	if(device->arguments.address_profile_4_4_16384 != NULL) {
		hipFree(device->arguments.address_profile_4_4_16384);
		device->arguments.address_profile_4_4_16384 = NULL;
	}

	if(device->arguments.segments_profile_4_4_16384 != NULL) {
		hipFree(device->arguments.segments_profile_4_4_16384);
		device->arguments.segments_profile_4_4_16384 = NULL;
	}

    if(device->arguments.memory_chunk_0 != NULL) {
        hipFree(device->arguments.memory_chunk_0);
        device->arguments.memory_chunk_0 = NULL;
    }

    if(device->arguments.memory_chunk_1 != NULL) {
        hipFree(device->arguments.memory_chunk_1);
        device->arguments.memory_chunk_1 = NULL;
    }

    if(device->arguments.memory_chunk_2 != NULL) {
        hipFree(device->arguments.memory_chunk_2);
        device->arguments.memory_chunk_2 = NULL;
    }

    if(device->arguments.memory_chunk_3 != NULL) {
        hipFree(device->arguments.memory_chunk_3);
        device->arguments.memory_chunk_3 = NULL;
    }

    if(device->arguments.memory_chunk_4 != NULL) {
        hipFree(device->arguments.memory_chunk_4);
        device->arguments.memory_chunk_4 = NULL;
    }

    if(device->arguments.memory_chunk_5 != NULL) {
        hipFree(device->arguments.memory_chunk_5);
        device->arguments.memory_chunk_5 = NULL;
    }

	if(device->arguments.seed_memory != NULL) {
		for(int i=0;i<2;i++) {
			if(device->arguments.seed_memory[i] != NULL)
				hipFree(device->arguments.seed_memory[i]);
			device->arguments.seed_memory[i] = NULL;
		}
	}

	if(device->arguments.out_memory != NULL) {
		for(int i=0;i<2;i++) {
			if(device->arguments.out_memory[i] != NULL)
				hipFree(device->arguments.out_memory[i]);
			device->arguments.out_memory[i] = NULL;
		}
	}

	if(device->arguments.host_seed_memory != NULL) {
		for(int i=0;i<2;i++) {
			if(device->arguments.host_seed_memory[i] != NULL)
				hipHostFree(device->arguments.host_seed_memory[i]);
			device->arguments.host_seed_memory[i] = NULL;
		}
	}

	hipDeviceReset();
}

void *cuda_kernel_filler(void *memory, int threads, argon2profile *profile, void *user_data) {
	cuda_gpumgmt_thread_data *gpumgmt_thread = (cuda_gpumgmt_thread_data *)user_data;
	cuda_device_info *device = gpumgmt_thread->device;
	hipStream_t stream = (hipStream_t)gpumgmt_thread->device_data;

	int mem_seed_count = profile->thr_cost;
	size_t work_items;

	uint32_t memsize;
	uint32_t parallelism;

	if(strcmp(profile->profile_name, "1_1_524288") == 0) {
		memsize = (uint32_t)argon2profile_1_1_524288.memsize;
		parallelism = argon2profile_1_1_524288.thr_cost;
	}
	else {
		memsize = (uint32_t)argon2profile_4_4_16384.memsize;
		parallelism = argon2profile_4_4_16384.thr_cost;
	}
	work_items = KERNEL_WORKGROUP_SIZE * parallelism;

	gpumgmt_thread->lock();

	device->error = hipMemcpyAsync(device->arguments.seed_memory[gpumgmt_thread->thread_id], memory, threads * 2 * mem_seed_count * ARGON2_BLOCK_SIZE, hipMemcpyHostToDevice, stream);
	if (device->error != hipSuccess) {
		device->error_message = "Error writing to gpu memory.";
		gpumgmt_thread->unlock();
		return NULL;
	}

	if(parallelism == 1) {
		fill_blocks_cpu<<<threads, work_items, 0, stream>>>((uint32_t*)device->arguments.memory_chunk_0,
                (uint32_t*)device->arguments.memory_chunk_1,
                (uint32_t*)device->arguments.memory_chunk_2,
                (uint32_t*)device->arguments.memory_chunk_3,
                (uint32_t*)device->arguments.memory_chunk_4,
                (uint32_t*)device->arguments.memory_chunk_5,
				device->arguments.seed_memory[gpumgmt_thread->thread_id],
				device->arguments.out_memory[gpumgmt_thread->thread_id],
				device->arguments.address_profile_1_1_524288,
				memsize, device->profile_info.threads_per_chunk_profile_1_1_524288, gpumgmt_thread->threads_profile_1_1_524288_idx);
	}
	else {
		fill_blocks_gpu<<<threads, work_items, 0, stream>>> ((uint32_t*)device->arguments.memory_chunk_0,
                (uint32_t*)device->arguments.memory_chunk_1,
                (uint32_t*)device->arguments.memory_chunk_2,
                (uint32_t*)device->arguments.memory_chunk_3,
                (uint32_t*)device->arguments.memory_chunk_4,
                (uint32_t*)device->arguments.memory_chunk_5,
				device->arguments.seed_memory[gpumgmt_thread->thread_id],
				device->arguments.out_memory[gpumgmt_thread->thread_id],
				device->arguments.address_profile_4_4_16384,
				device->arguments.segments_profile_4_4_16384,
				memsize, device->profile_info.threads_per_chunk_profile_4_4_16384, gpumgmt_thread->threads_profile_4_4_16384_idx);
	}

	device->error = hipMemcpyAsync(memory, device->arguments.out_memory[gpumgmt_thread->thread_id], threads * 2 * mem_seed_count * ARGON2_BLOCK_SIZE, hipMemcpyDeviceToHost, stream);
	if (device->error != hipSuccess) {
		device->error_message = "Error reading gpu memory.";
		gpumgmt_thread->unlock();
		return NULL;
	}

	while(hipStreamQuery(stream) != hipSuccess) {
		this_thread::sleep_for(chrono::milliseconds(10));
		continue;
	}

	gpumgmt_thread->unlock();

	return memory;
}
